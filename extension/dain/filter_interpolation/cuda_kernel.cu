#include "hip/hip_runtime.h"
#include <stdio.h>

#include "cuda_kernel.cuh"


#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>


#define min(a,b) ((a<b)?(a):(b))
#define max(a,b) ((a>b)?(a):(b))

#define DEBUG (0)
#ifndef BLOCKDIMX
#define BLOCKDIMX (32)
#endif
#ifndef BLOCKDIMY
#define BLOCKDIMY (16)
#endif
using at::Half;




//forward path of our layer
template <typename scalar_t>
__global__ void FilterInterpolationLayer_gpu_forward_kernelfunc(
		const int nElement,
		const int w, 		const int h, 		const int channel, const int filter_size,

		const int input1_b_stride, const int input1_c_stride, const int input1_h_stride, const int input1_w_stride,
		const int input2_b_stride, const int input2_c_stride, const int input2_h_stride, const int input2_w_stride,
		const int input3_b_stride, const int input3_c_stride, const int input3_h_stride, const int input3_w_stride,

		const scalar_t* __restrict__    input1,    		const scalar_t* __restrict__    input2,    	const scalar_t* __restrict__    input3, 	scalar_t*   output

		)
{

	//blockIdx.z : batch index from 0~B-1
	//blockIdx.y : height patch index from ceil(h/16)
	//blockIdx.x : width patch index from ceil(w/32)

	//threadidx.x: width index 0~31
	//threadIdx.y: height index 0~15
	//threadIdx.z: Not used

	//only use one dimensioon of the grid and block
	const int w_i = blockIdx.x * blockDim.x + threadIdx.x;
	const int h_i = blockIdx.y * blockDim.y + threadIdx.y;
	const bool withinXbounds = w_i < w;
	const bool withinYbounds = h_i < h;

	const int batch_i = blockIdx.z;
	const int off = batch_i * input1_b_stride;


	//    __syncthreads();
//	const float fillvalue =0.0f;

	if( withinXbounds && withinYbounds) {

		float fx = input2[batch_i * input2_b_stride + 0 * input2_c_stride + h_i * input2_h_stride + w_i  ];
		float fy = input2[batch_i * input2_b_stride + 1 * input2_c_stride + h_i * input2_h_stride + w_i  ];

		float x2 = (float)(w_i) + fx;
		float y2 = (float)(h_i) + fy;


		if(x2 >= 0.0f && y2 >=0.0f && x2 <= (float)(w -1) && y2 <= (float)(h-1)
            && fabs(fx) < (float)(w)/2.0f && fabs(fy) < (float)(h)/2.0f){
			int ix2_L = int(x2) + 1 - (int)(filter_size / 2);
			int iy2_T = int(y2) + 1 - (int)(filter_size / 2);
			int ix2_R = ix2_L + filter_size;
			int iy2_B = iy2_T + filter_size;

            float alpha = x2 - (int)(x2);
            float beta = y2 - (int)(y2);


			//TODO: here is a bug that if the iy2_B or ix2_R gets out of the border, than there is no enough pixels to warp the target one.
			for (int c_i = 0 ; c_i < channel ; c_i++){

                float TL = 0.0f;
                for(int filter_j = iy2_T; filter_j <= (int)(y2); filter_j ++){
                    int _filter_j = min(max(0, filter_j), h - 1);
                    for( int filter_i = ix2_L; filter_i <= (int) ( x2) ; filter_i ++ ){
                    int _filter_i = min(max(0, filter_i ), w - 1);
                    TL += input1[off + c_i *  input1_c_stride +  _filter_j * input1_h_stride + _filter_i ] *
							input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i] ;
                    }
                }

                float TR = 0.0f;
                for (int filter_j = iy2_T; filter_j <= (int) (y2); filter_j ++ ){
                    int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                for (int filter_i =  (int) (x2) + 1 ; filter_i < ix2_R; filter_i ++ ){
                    int _filter_i = min(max(0, filter_i),w - 1);// only used for input1
                    TR += input1 [off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
                        input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i];
                }
                }

                float BL = 0.0f;
                for (int filter_j = (int) (y2) + 1; filter_j < iy2_B; filter_j ++ ){
                    int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                for (int filter_i = ix2_L; filter_i <= (int) (x2); filter_i ++ ){
                    int _filter_i = min(max(0, filter_i),w - 1);// only used for input1
                    BL += input1 [off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
                        input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i];
                }
                }

                float BR = 0.0f;
                for (int filter_j = (int) (y2) + 1; filter_j < iy2_B; filter_j ++ ){
                    int _filter_j = min(max(0, filter_j),h - 1); // only used for input1
                for (int filter_i = (int) (x2) + 1; filter_i < ix2_R; filter_i ++ ){
                    int _filter_i = min(max(0, filter_i),w - 1);// only used for input1
                    BR += input1 [off + c_i * input1_c_stride + _filter_j * input1_h_stride + _filter_i] *
                        input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i];
                }
                }

                output[off + c_i * input1_c_stride + h_i * input1_h_stride + w_i ] =
                            (1-alpha)*(1-beta)*TL +
							alpha*(1-beta)*TR +
							(1-alpha)*beta*BL +
							alpha*beta*BR;

//					for( int filter_i = ix2_L; filter_i < ix2_R ; filter_i ++ ){
//						int _filter_i = min(max(0, filter_i),w - 1);
//						output[off + c_i * input1_c_stride + h_i * input1_h_stride + w_i ] +=
//							input1[off + c_i *  input1_c_stride +  _filter_j * input1_h_stride + _filter_i ] *
//							input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i] *
////							exp( -(fabs((float) filter_j - y2) + fabs((float) filter_i - x2)) / (float)(filter_size)); // the distance weight
//							exp( -(fabs((float) filter_j - y2) + fabs((float) filter_i - x2)) ); // the distance weight
//
////							if(w_i == 141 && h_i == 316 && c_i == 0 ){
////printf("gpu: %f, %f,%f,%f\n",input1[off + c_i *  input1_c_stride +  _filter_j * input1_h_stride + _filter_i ] ,
////input3 [batch_i * input3_b_stride + ((filter_j - iy2_T) * filter_size + (filter_i - ix2_L)) * input3_c_stride + h_i * input3_h_stride + w_i],
////exp( -(fabs((float) filter_j - y2) + fabs((float) filter_i - x2)) / (float)(filter_size)),
////output[off + c_i * input1_c_stride + h_i * input1_h_stride + w_i ]
//// );
////}
//
//					}
//				}
			}
		} else{
			//the warping data is out of range, we fill it with zeros
			for(int c_i = 0 ;  c_i < channel; c_i ++){
				output[off + c_i * input1_c_stride + h_i * input1_h_stride + w_i] = input1[off + c_i* input1_c_stride+ h_i * input1_h_stride + w_i];
			}
		}
	}
	return ;

}


int FilterInterpolationLayer_gpu_forward_kernel(
		hipStream_t stream,
		const int nElement,
		const int w, 		const int h, 		const int channel, 		const int batch, const  int filter_size,

		const int input1_b_stride, const int input1_c_stride, const int input1_h_stride, const int input1_w_stride,
		const int input2_b_stride, const int input2_c_stride, const int input2_h_stride, const int input2_w_stride,
		const int input3_b_stride, const int input3_c_stride, const int input3_h_stride, const int input3_w_stride,

		at::Tensor&  input1,    		at::Tensor&  input2,    	at::Tensor&  input3, 	at::Tensor&  output

		)
{
	int error = 1 ;

	dim3 grid;
	dim3 block;


	//		blockthread = 128;
	//the threadIdx.x is sheduled first, then threadIdx.y, threadIdx.z
	//the three channels are processsed in one kernel
	block  = dim3(BLOCKDIMX,BLOCKDIMY,1);
	grid = dim3( (w + BLOCKDIMX - 1)/ BLOCKDIMX, (h + BLOCKDIMY - 1) / BLOCKDIMY, batch);
    if(BLOCKDIMX != 32 || BLOCKDIMY != 16||DEBUG)
        printf("BLOCKDIMX revised to %d, BLOCKDIMY revised to %d \n", BLOCKDIMX,BLOCKDIMY);
	//extract the data of CudaTensor and use kernel to calculate.
		AT_DISPATCH_FLOATING_TYPES(input1.type(), "DepthFlowProjection_gpu_backward", ([&] {
FilterInterpolationLayer_gpu_forward_kernelfunc<<<grid,block,0, stream >>>(
			nElement, //to let the nummous
			w,h,channel,filter_size,
			input1_b_stride,input1_c_stride,input1_h_stride,input1_w_stride,
			input2_b_stride,input2_c_stride,input2_h_stride,input2_w_stride,
			input3_b_stride,input3_c_stride,input3_h_stride,input3_w_stride,

			input1.data<scalar_t>(),input2.data<scalar_t>(),input3.data<scalar_t>(), output.data<scalar_t>()
			);
 					}));

	//			THCudaCheck(hipGetLastError());
	hipError_t err = hipGetLastError();

	if (err != hipSuccess) {
		printf("gpuerror in BilinearSampler.updateOutput: %s\n", hipGetErrorString(err));
		//THError("aborting");
		return error;
	}

	error = 0;
	return error;

}
